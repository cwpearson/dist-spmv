#include "hip/hip_runtime.h"
#include <mpi.h>

#include <nvToolsExt.h>
// #include <hip/hip_runtime_api.h>

#include <vector>
#include <string>
#include <stdexcept>
#include <algorithm>
#include <iostream>

#include "cuda_runtime.hpp"

#define STRINGIFY(x) #x
#define TOSTRING(x) STRINGIFY(x)
#define AT __FILE__ ":" TOSTRING(__LINE__) 

//#define VIEW_CHECK_BOUNDS

template<typename ForwardIt>
void shift_left(ForwardIt first, ForwardIt last, size_t n) {
    while(first != last) {
        *(first-n) = *first;
        ++first;
    }
}

enum Tag : int {
    row_ptr,
    col_ind,
    val,
    x,
    num_cols
};

enum class Where {
    host,
    device
};

template <Where where, typename T>
class Array;


// A non-owning view of data
template <typename T>
struct ArrayView
{
    T *data_;
    int64_t size_;
    public:
    ArrayView() : data_(nullptr), size_(0){}
    ArrayView(const ArrayView &other) = default;
    ArrayView(ArrayView &&other) = default;
    ArrayView &operator=(const ArrayView &rhs) = default;

    __host__ __device__ int64_t size() const { return size_; }

    __host__ __device__ const T &operator()(int64_t i) const {
#ifdef VIEW_CHECK_BOUNDS
        if (i < 0) {
            printf("ERR: i < 0: %d\n", i);
        }
        if (i >= size_) {
            printf("ERR: i > size_: %d > %ld\n", i, size_);
        }
#endif
        return data_[i];
    }
    __host__ __device__ T &operator()(int64_t i) {
        return data_[i];
    }
};

/* device array
*/
template<typename T> class Array<Where::device, T>
{
public:

    // array owns the data in this view
    ArrayView<T> view_;
public:
    Array() = default;
    Array(const size_t n) {
        resize(n);
    }
    Array(const Array &other) = delete;
    Array(Array &&other) : view_(other.view_) {
        // view is non-owning, so have to clear other
        other.view_.data_ = nullptr;
        other.view_.size_ = 0;
    }

    Array(const std::vector<T> &v) {
        set_from(v);
    }

    ~Array() {
        CUDA_RUNTIME(hipFree(view_.data_));
        view_.data_ = nullptr;
        view_.size_ = 0;
    }
    int64_t size() const { 
        return view_.size(); }

    ArrayView<T> view() const {
        return view_; // copy of internal view
    }

    operator std::vector<T>() const {
        std::vector<T> v(size());
        CUDA_RUNTIME(hipMemcpy(v.data(), view_.data_, size() * sizeof(T), hipMemcpyDeviceToHost));
        return v;
    }

    void set_from(const std::vector<T> &rhs, hipStream_t stream = 0) {
        resize(rhs.size());
        CUDA_RUNTIME(hipMemcpyAsync(view_.data_, rhs.data(), view_.size_ * sizeof(T), hipMemcpyHostToDevice, stream));
    }

    void set_from(const Array<Where::host, T> &rhs, hipStream_t stream = 0) {
        resize(rhs.size());
        CUDA_RUNTIME(hipMemcpyAsync(view_.data_, rhs.data(), view_.size_ * sizeof(T), hipMemcpyHostToDevice, stream));
    }

    // any change destroys all data
    void resize(size_t n) {
        if (size() != n) {
            view_.size_ = n;
            CUDA_RUNTIME(hipFree(view_.data_));
            CUDA_RUNTIME(hipMalloc(&view_.data_, view_.size_ * sizeof(T)));
        }
    }

};


/* host array
*/
template<typename T> class Array<Where::host, T>
{
public:

    // array owns the data in this view
    ArrayView<T> view_;
public:
    Array() = default;
    Array(const size_t n, const T &val) {
        resize(n);
        for (size_t i = 0; i < n; ++i) {
            view_(i) = val;
        }
    }
    Array(const Array &other) = delete;
    Array(Array &&other) : view_(other.view_) {
        // view is non-owning, so have to clear other
        other.view_.data_ = nullptr;
        other.view_.size_ = 0;
    }

    ~Array() {
        CUDA_RUNTIME(hipHostFree(view_.data_));
        view_.data_ = nullptr;
        view_.size_ = 0;
    }
    int64_t size() const { 
        return view_.size(); }

    ArrayView<T> view() const {
        return view_; // copy of internal view
    }

    // any change destroys all data
    void resize(size_t n) {
        if (size() != n) {
            view_.size_ = n;
            CUDA_RUNTIME(hipHostFree(view_.data_));
            CUDA_RUNTIME(hipHostAlloc(&view_.data_, view_.size_ * sizeof(T), hipHostMallocDefault));
        }
    }

    const T* data() const {
        return view_.data_;
    }
    T* data() {
        return view_.data_;
    }

};


class CooMat {
public:


    struct Entry {
        int i;
        int j;
        float e;

        Entry(int _i, int _j, int _e) : i(_i), j(_j), e(_e) {}

        static bool by_ij(const Entry &a, const Entry &b) {
            if (a.i < b.i) {
                return true;
            } else if (a.i > b.i) {
                return false;
            } else {
                return a.j < b.j;
            }
        }

        static bool same_ij(const Entry &a, const Entry &b) {
            return a.i == b.i && a.j == b.j;
        }
    };

private:

    // sorted during construction
    std::vector<Entry> data_;
    int64_t numRows_;
    int64_t numCols_;

public:
    CooMat(int m, int n) : numRows_(m), numCols_(n) {}
    const std::vector<Entry> &entries() const {return data_;}
    void push_back(int i, int j, int e) {
        data_.push_back(Entry(i,j,e));  
    }

    void sort() {
        std::sort(data_.begin(), data_.end(), Entry::by_ij);
    }

    void remove_duplicates() {
        std::sort(data_.begin(), data_.end(), Entry::by_ij);
        std::unique(data_.begin(), data_.end(), Entry::same_ij);
    }

    int64_t num_rows() const {return numRows_;}
    int64_t num_cols() const {return numCols_;}
    int64_t nnz() const {return data_.size();}
};

template <Where where>
class CsrMat {
public:
    CsrMat();
    int64_t nnz() const;
    int64_t num_rows() const;
};
template<> class CsrMat<Where::host>;
template<> class CsrMat<Where::device>;

/* host sparse matrix */
template<> class CsrMat<Where::host>
{
    friend class CsrMat<Where::device>; // device can see inside
    std::vector<int> rowPtr_;
    std::vector<int> colInd_;
    std::vector<float> val_;
    int64_t numCols_;

public:
    CsrMat() = default;
    CsrMat(int numRows, int numCols, int nnz) : rowPtr_(numRows+1), colInd_(nnz), val_(nnz), numCols_(numCols) {}

    CsrMat(const CooMat &coo) : numCols_(coo.num_cols()) {
        for (auto &e : coo.entries()) {
            while (rowPtr_.size() <= e.i) {
                rowPtr_.push_back(colInd_.size());
            }
            colInd_.push_back(e.j);
            val_.push_back(e.e);
        }
        while (rowPtr_.size() < coo.num_rows()+1){
            rowPtr_.push_back(colInd_.size());
        }
    }

    int64_t num_rows() const {
      if (rowPtr_.size() <= 1) {
        return 0; 
      } else { 
        return rowPtr_.size() - 1;
      }
    }

    int64_t num_cols() const {
        return numCols_;
      }

    int64_t nnz() const {
        if (colInd_.size() != val_.size()) {
            throw std::logic_error("bad invariant");
        }
        return colInd_.size();
    }

    const int &row_ptr(int64_t i) const {
        return rowPtr_[i];
    }
    const int &col_ind(int64_t i) const {
        return colInd_[i];
    }
    const float &val(int64_t i) const {
        return val_[i];
    }

    const int *row_ptr() const {return rowPtr_.data(); }
    int *row_ptr() {return rowPtr_.data(); }
    const int *col_ind() const {return colInd_.data(); }
    int *col_ind() {return colInd_.data(); }
    const float *val() const {return val_.data(); }
    float *val() {return val_.data(); }

    /* keep rows [rowStart, rowEnd)
    */
    void retain_rows(int rowStart, int rowEnd) {
        
        if (0 == rowEnd) {
            throw std::logic_error("unimplemented");
        }
        // erase rows after
        // dont want to keep rowEnd, so rowEnd points to end of rowEnd-1
        std::cerr << "rowPtr_ = rowPtr[:" << rowEnd+1 << "]\n";
        rowPtr_.resize(rowEnd+1);
        std::cerr << "resize entries to " << rowPtr_.back() << "\n";
        colInd_.resize(rowPtr_.back());
        val_.resize(rowPtr_.back());

        // erase early row pointers
        std::cerr << "rowPtr <<= " << rowStart << "\n";
        shift_left(rowPtr_.begin()+rowStart, rowPtr_.end(), rowStart);
        std::cerr << "resize rowPtr to " << rowEnd - rowStart+1 << "\n";
        rowPtr_.resize(rowEnd-rowStart+1);

        const int off = rowPtr_[0];
        // erase entries for first rows
        std::cerr << "entries <<= " << off << "\n";
        shift_left(colInd_.begin()+off, colInd_.end(), off);
        shift_left(val_.begin()+off, val_.end(), off);

        // adjust row pointer offset
        std::cerr << "subtract rowPtrs by " << off << "\n";
        for (auto &e : rowPtr_) {
            e -= off;
        }

        // resize entries
        std::cerr << "resize entries to " << rowPtr_.back() << "\n";
        colInd_.resize(rowPtr_.back());
        val_.resize(rowPtr_.back());
    }

};

/* device sparse matrix
*/
template<> class CsrMat<Where::device>
{
    Array<Where::device, int> rowPtr_;
    Array<Where::device, int> colInd_;
    Array<Where::device, float> val_;

public:

    struct View {
        ArrayView<int> rowPtr_;
        ArrayView<int> colInd_;
        ArrayView<float> val_;

        __device__ int num_rows() const {
            if (rowPtr_.size() > 0) {
                return rowPtr_.size() - 1;
            } else {
                return 0;
            }
        }

        __device__ const int &row_ptr(int64_t i) const {
            return rowPtr_(i);
        }

        __device__ const int &col_ind(int64_t i) const {
            return colInd_(i);
        }

        __device__ const float &val(int64_t i) const {
            return val_(i);
        }

    };

    CsrMat() = delete;
    CsrMat(CsrMat &&other) = delete;
    CsrMat(const CsrMat &other) = delete;

    // create device matrix from host
    CsrMat(const CsrMat<Where::host> &m) : 
        rowPtr_(m.rowPtr_), colInd_(m.colInd_), val_(m.val_) {
        if (colInd_.size() != val_.size()) {
            throw std::logic_error("bad invariant");
        }
    }
    ~CsrMat() {
    }
    int64_t num_rows() const {
        if (rowPtr_.size() <= 1) {
            return 0; 
          } else { 
            return rowPtr_.size() - 1;
          }
    }
  
    int64_t nnz() const {
        return colInd_.size();
    }

    View view() const {
        View v;
        v.rowPtr_ = rowPtr_.view();
        v.colInd_ = colInd_.view();
        v.val_ = val_.view();
        return v;
    }

};




// mxn random matrix with nnz
CsrMat<Where::host> random_matrix(const int64_t m, const int64_t n, const int64_t nnz) {

    if (m * n < nnz) {
        throw std::logic_error(AT);
    }

    CooMat coo(m,n);
    while(coo.nnz() < nnz) {

        int64_t toPush = nnz - coo.nnz();
        std::cerr << "adding " << toPush << " non-zeros\n";
        for (int64_t _ = 0; _ < toPush; ++_) {
            int r = rand() % m;
            int c = rand() % n;
            float e = 1.0;
            coo.push_back(r, c, e);
        }
        std::cerr << "removing duplicate non-zeros\n";
        coo.remove_duplicates();
    }
    coo.sort();
    std::cerr << "coo: " << coo.num_rows() << "x" << coo.num_cols() << "\n";
    CsrMat<Where::host> csr(coo);
    std::cerr << "csr: " << csr.num_rows() << "x" << csr.num_cols() << " w/ " << csr.nnz() << "\n";
    return csr;
};

// nxn diagonal matrix with bandwidth b
CsrMat<Where::host> random_band_matrix(const int64_t n, const int64_t bw, const int64_t nnz) {

    CooMat coo(n,n);
    while(coo.nnz() < nnz) {

        int64_t toPush = nnz - coo.nnz();
        std::cerr << "adding " << toPush << " non-zeros\n";
        for (int64_t _ = 0; _ < toPush; ++_) {
            int r = rand() % n; // random row

            // column in the band
            int lb = r - bw;
            int ub = r + bw + 1;
            int64_t c = rand() % (ub - lb) + lb;
            if (c < 0 || c >= n) {
                continue; // don't over-weight first or last column
            }
            
            float e = 1.0;
            coo.push_back(r, c, e);
        }
        std::cerr << "removing duplicate non-zeros\n";
        coo.remove_duplicates();
    }
    coo.sort();
    std::cerr << "coo: " << coo.num_rows() << "x" << coo.num_cols() << "\n";
    CsrMat<Where::host> csr(coo);
    std::cerr << "csr: " << csr.num_rows() << "x" << csr.num_cols() << " w/ " << csr.nnz() << "\n";
    return csr;
};

std::vector<float> random_vector(const int64_t n) {
    return std::vector<float>(n, 1.0);
}

Array<Where::host, float> random_array(const int64_t n) {
    return Array<Where::host, float>(n, 1.0);
}

struct Range {
    int lb;
    int ub;
};

/* get the ith part of splitting domain in to n pieces
*/
Range get_partition(const int domain, const int i, const int n) {
    int div = domain / n;
    int rem = domain % n;

    int lb, ub;

    if (i < rem) {
        lb = i * (div+1);
        ub = lb + (div+1);
    } else {
        lb = rem * (div+1) + (i-rem) * div;
        ub = lb + div;
    }
    return Range{.lb=lb, .ub=ub};
}

std::vector<CsrMat<Where::host>> part_by_rows(const CsrMat<Where::host> &m, const int parts) {

    std::vector<CsrMat<Where::host>> mats;

    for (int p = 0; p < parts; ++p) {
        Range range = get_partition(m.num_rows(), p, parts);
        std::cerr << "matrix part " << p << " has " << range.ub-range.lb << " rows\n";
        CsrMat<Where::host> part(m);
        part.retain_rows(range.lb, range.ub);
        mats.push_back(part);
    }

    return mats;
}

struct DistMat {
    CsrMat<Where::host> local;
    CsrMat<Where::host> remote;
};

DistMat split_local_remote(const CsrMat<Where::host> &m, MPI_Comm comm) {
    int rank = 0;
    int size = 1;
    MPI_Comm_rank(comm, &rank);
    MPI_Comm_size(comm, &size);

    // which rows of x are local
    Range localRange = get_partition(m.num_cols(), rank, size);

    // build two matrices, local gets local non-zeros, remote gets remote non-zeros
    CooMat local(m.num_rows(), m.num_cols()), remote(m.num_rows(), m.num_cols());

    for (int r = 0; r < m.num_rows(); ++r) {
        for (int ci = m.row_ptr(r); ci < m.row_ptr(r+1); ++ci) {
            int c = m.col_ind(ci);
            float v = m.val(ci);

            if (c >= localRange.lb && c < localRange.ub) {
                local.push_back(r,c,v);
            } else {
                remote.push_back(r,c,v);
            }

        }
    }

    return DistMat {
        .local=local,
        .remote=remote
    };

}


std::vector<std::vector<float>> part_by_rows(const std::vector<float> &x, const int parts) {
    std::vector<std::vector<float>> xs;

    for (int p = 0; p < parts; ++p) {
        Range range = get_partition(x.size(), p, parts);
        std::cerr << "vector part " << p << " will have " << range.ub-range.lb << " rows\n";
        std::vector<float> part(x.begin()+range.lb, x.begin()+range.ub);
        xs.push_back(part);
    }

    if (xs.size() != parts) {
        throw std::logic_error("line " STRINGIFY(__LINE__));
    }
    return xs;
}

int send_matrix(int dst, int src, CsrMat<Where::host> &&m, MPI_Comm comm) {

    MPI_Request reqs[4];

    int numCols = m.num_cols();
    MPI_Isend(&numCols, 1, MPI_INT, dst, Tag::num_cols, comm, &reqs[0]);
    MPI_Isend(m.row_ptr(), m.num_rows()+1, MPI_INT, dst, Tag::row_ptr, comm, &reqs[1]);
    MPI_Isend(m.col_ind(), m.nnz(), MPI_INT, dst, Tag::col_ind, comm, &reqs[2]);
    MPI_Isend(m.val(), m.nnz(), MPI_FLOAT, dst, Tag::val, comm, &reqs[3]);
    MPI_Waitall(4, reqs, MPI_STATUSES_IGNORE);

    return 0;
}

CsrMat<Where::host> receive_matrix(int dst, int src, MPI_Comm comm) {

    int numCols;
    MPI_Recv(&numCols, 1, MPI_INT, 0, Tag::num_cols, comm, MPI_STATUS_IGNORE);

    // probe for number of rows
    MPI_Status stat;
    MPI_Probe(0, Tag::row_ptr, comm, &stat);
    int numRows;
    MPI_Get_count(&stat, MPI_INT, &numRows);
    if (numRows > 0) {
        --numRows;
    }

    // probe for nnz
    MPI_Probe(0, Tag::col_ind, comm, &stat);
    int nnz;
    MPI_Get_count(&stat, MPI_INT, &nnz);

    std::cerr << "recv " << numRows << "x" << numCols << " w/ " << nnz << "\n";
    CsrMat<Where::host> csr(numRows, numCols, nnz);

    // receive actual data into matrix
    MPI_Recv(csr.row_ptr(), numRows+1, MPI_INT, 0, Tag::row_ptr, comm, MPI_STATUS_IGNORE);
    MPI_Recv(csr.col_ind(), nnz, MPI_INT, 0, Tag::col_ind, comm, MPI_STATUS_IGNORE);
    MPI_Recv(csr.val(), nnz, MPI_FLOAT, 0, Tag::val, comm, MPI_STATUS_IGNORE);

    return csr;
}

int send_x(int dst, int src, std::vector<float> &&v, MPI_Comm comm) {
    MPI_Send(v.data(), v.size(), MPI_FLOAT, dst, Tag::x, comm);
    return 0;
}

/* recv some amount of data, and put it in the right place
   in a full x
*/
std::vector<float> receive_x(const int n, const int dst, int src, MPI_Comm comm) {
    int rank = 0;
    int size = 1;
    MPI_Comm_rank(comm, &rank);
    MPI_Comm_size(comm, &size);

    // which rows of x are local
    Range local = get_partition(n, rank, size);

    // probe for size
    MPI_Status stat;
    MPI_Probe(0, Tag::x, comm, &stat);
    int sz;
    MPI_Get_count(&stat, MPI_INT, &sz);
    if (sz != local.ub-local.lb) {
        throw std::logic_error(AT);
    }

    std::cerr << "recv " << sz << " x entries into offset " << local.lb << "\n";
    std::vector<float> x(n);
    MPI_Recv(x.data() + local.lb, sz, MPI_FLOAT, 0, Tag::x, comm, MPI_STATUS_IGNORE);

    return x;
}

enum class ProductConfig {
    MODIFY, // b += 
    SET     // b =
};

/* Ax=b
*/
__global__ void spmv(ArrayView<float> b,
     const CsrMat<Where::device>::View A,
      const ArrayView<float> x,
      const ProductConfig pc
    ) {
    // one thread per row
    for (int r = blockDim.x * blockIdx.x + threadIdx.x; r < A.num_rows(); r += blockDim.x * gridDim.x) {
        float acc = 0;
        for (int ci = A.row_ptr(r); ci < A.row_ptr(r+1); ++ci) {
            int c = A.col_ind(ci);
            acc += A.val(ci) * x(c);
        }
        if (ProductConfig::SET == pc) {
            b(r) = acc;
        } else {
            b(r) += acc;
        }
    }
}

// z += a
__global__ void vector_add(ArrayView<float> z, const ArrayView<float> a) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < z.size(); i += blockDim.x * gridDim.x) {
        z(i) += a(i);
    }
}

int main (int argc, char **argv) {

    MPI_Init(&argc, &argv);

    int rank = 0;
    int size = 1;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);
  
    std::cerr << "get a gpu...\n";
    CUDA_RUNTIME(hipSetDevice(rank % 4));
    CUDA_RUNTIME(hipFree(0));
    std::cerr << "barrier...\n";
    MPI_Barrier(MPI_COMM_WORLD);

    // int64_t m = 150000;
    // int64_t n = 150000;
    // int64_t nnz = 11000000;
    // or
    int64_t m = 150000;
    int64_t n = m;
    int64_t bw = m/size; // ~50% local vs remote non-zeros for most ranks
    int64_t nnz = 11000000;

    CsrMat<Where::host> lA; // "local A"

    // generate and distribute A
    if (0 == rank) {
        std::cerr << "generate matrix\n";
        // lA = random_matrix(m, n, nnz);
        lA = random_band_matrix(m, bw, nnz);
        std::cerr << "partition matrix\n";
        std::vector<CsrMat<Where::host>> As = part_by_rows(lA, size);
        for (size_t dst = 1; dst < size; ++dst) {
            std::cerr << "send A to " << dst << "\n";
            send_matrix(dst, 0, std::move(As[dst]), MPI_COMM_WORLD);
        }
        lA = As[rank];
    } else {
        std::cerr << "recv A at " << rank << "\n";
        lA = receive_matrix(rank, 0, MPI_COMM_WORLD);
    }

    // each rank has a dense x. each rank owns part of it,
    // but it doesn't matter what the entries are
    Array<Where::host, float> lx = random_array(n); // "local x"
    std::cerr << "local X: " << lx.size() << "\n";
    std::cerr << "copy x to device\n";
    Array<Where::device, float> lxd(lx.size()), rxd(lx.size()); // "local/remote x device"

    // get a local and remote split of A
    std::cerr << "split local/remote A\n";
    CsrMat<Where::host> rA, A(lA);
    {
        DistMat d = split_local_remote(lA, MPI_COMM_WORLD);
        lA = d.local;
        rA = d.remote;
    }
    std::cerr << "A:        " << A.num_rows() << "x" << A.num_cols() << " w/ " << A.nnz() << "\n";
    std::cerr << "local A:  " << lA.num_rows() << "x" << lA.num_cols() << " w/ " << lA.nnz() << "\n";
    std::cerr << "remote A: " << rA.num_rows() << "x" << rA.num_cols() << " w/ " << rA.nnz() << "\n";

    std::cerr << "Copy A to GPU\n";
    CsrMat<Where::device> Ad(A), lAd(lA), rAd(rA);


    // Product vector size is same as local rows of A
    std::vector<float> b(lA.num_rows(), 0);
    std::cerr << "Copy b to GPU\n";
    Array<Where::device, float> lbd(b), rbd(b); // "local b device, remote b device"


    // plan allgather of remote x data
    std::cerr << "plan allgather xs\n";
    std::vector<int> recvcounts;
    std::vector<int> displs;
    for (int i = 0; i < size; ++i) {
        Range r = get_partition(lx.size(), i, size);
        recvcounts.push_back(r.ub-r.lb);
        if (displs.empty()) {
            displs.push_back(0);
        } else {
            displs.push_back(displs.back() + recvcounts.back());
        }
    }

    int loPrio, hiPrio;
    CUDA_RUNTIME(hipDeviceGetStreamPriorityRange (&loPrio, &hiPrio));

    hipStream_t loS, hiS; // "lo/hi prio"
    CUDA_RUNTIME(hipStreamCreateWithPriority(&loS, hipStreamNonBlocking, hiPrio));
    CUDA_RUNTIME(hipStreamCreateWithPriority(&hiS, hipStreamNonBlocking, hiPrio));

    hipEvent_t event;
    CUDA_RUNTIME(hipEventCreateWithFlags(&event, hipEventDisableTiming));

    const int nIters = 30;
    std::vector<double> times(nIters);

    /* ===== multiply in one shot
    */

    // do spmv
    dim3 dimBlock(256);
    dim3 dimGrid(100);

    nvtxRangePush("one-shot");
    for (int i = 0; i < nIters; ++i) {
        MPI_Barrier(MPI_COMM_WORLD);
        double start = MPI_Wtime();

        // distribute x to ranks
        MPI_Allgatherv(lx.data() + displs[rank], recvcounts[rank], MPI_FLOAT, lx.data(), recvcounts.data(), displs.data(), MPI_FLOAT, MPI_COMM_WORLD);

        // copy x to GPU
        lxd.set_from(lx, hiS);

        spmv<<<dimGrid, dimBlock, 0, hiS>>>(lbd.view(), Ad.view(), lxd.view(), ProductConfig::SET);
        CUDA_RUNTIME(hipGetLastError());
        CUDA_RUNTIME(hipStreamSynchronize(hiS));
        times[i] = MPI_Wtime() - start;
    }
    nvtxRangePop(); // one-shot
    MPI_Allreduce(MPI_IN_PLACE, times.data(), times.size(), MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    if (0 == rank) {
        std::sort(times.begin(), times.end());
        std::cerr << times[times.size() / 2] << "\n";
    }


    /* ===== split local and remote
       multiply local, gather & multiply remote
       TODO: the separate add launch can be removed if it is ensured
       that The remote happens strictly after the local.
       It's a small false serialization, but if we're in the case
       where that matters, the launch overhead dominates anyway.
    */
    nvtxRangePush("local/remote");
    for (int i = 0; i < nIters; ++i) {

        MPI_Barrier(MPI_COMM_WORLD);
        double start = MPI_Wtime();

        // overlap MPI with CUDA kernel launch
        MPI_Request req;
        MPI_Iallgatherv(lx.data() + displs[rank], recvcounts[rank], MPI_FLOAT, lx.data(), recvcounts.data(), displs.data(), MPI_FLOAT, MPI_COMM_WORLD, &req);

        spmv<<<dimGrid, dimBlock, 0, hiS>>>(lbd.view(), lAd.view(), lxd.view(), ProductConfig::SET);
        CUDA_RUNTIME(hipGetLastError());

        MPI_Wait(&req, MPI_STATUS_IGNORE);

        rxd.set_from(lx, loS);
        
        // hiS blocks until transfer is done
        CUDA_RUNTIME(hipEventRecord(event, loS));
        CUDA_RUNTIME(hipStreamWaitEvent(hiS, event, 0));

        spmv<<<dimGrid, dimBlock, 0, hiS>>>(rbd.view(), rAd.view(), rxd.view(), ProductConfig::MODIFY);
        CUDA_RUNTIME(hipGetLastError());

        // all is done when hiS is done
        CUDA_RUNTIME(hipStreamSynchronize(hiS));
        times[i] = MPI_Wtime() - start;
    }
    nvtxRangePop(); // local/remote
    MPI_Allreduce(MPI_IN_PLACE, times.data(), times.size(), MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
    if (0 == rank) {
        std::sort(times.begin(), times.end());
        std::cerr << times[times.size() / 2] << "\n";
    }

    // maybe better to atomic add into result than doing separate kernel launch?

    CUDA_RUNTIME(hipStreamDestroy(loS));
    CUDA_RUNTIME(hipStreamDestroy(hiS));

    MPI_Finalize();

    return 0;
}